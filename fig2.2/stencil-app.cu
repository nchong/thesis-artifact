
#include <hip/hip_runtime.h>
__global__ void stencil(int *A, int *B, int radius, unsigned n) {
  unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;

  int sum = 0;
  for (int i=-radius; i<=radius; i++) {
    int idx = tid + i;
    if (0 <= idx && idx < n) {
      sum += A[idx];
    }
  }

  if (tid < n) B[tid] = sum;
}

void hostStencil(int *A, int *B, int radius, unsigned n) {
  // device copies of A and B
  int *d_A; int *d_B;

  // allocate arrays on device
  size_t sz = sizeof(int)*n;
  hipMalloc(&d_A, sz); hipMalloc(&d_B, sz);

  // copy input to device
  hipMemcpy(d_A, A, sz, hipMemcpyHostToDevice);

  // launch kernel
  stencil<<<1,n>>>(d_A, d_B, radius, n);

  // copy output from device
  hipMemcpy(B, d_B, sz, hipMemcpyDeviceToHost);

  // free allocated arrays
  hipFree(d_A); hipFree(d_B);
}

