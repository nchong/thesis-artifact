#include "hip/hip_runtime.h"
//--gridDim=[1] --blockDim=[4]

#define N 4

__global__ void saxpy(float a, float *x, float *y) {
  unsigned tid = threadIdx.x;

  for (int i=0;
       __invariant((0 <= i) & (i <= N)),
       __invariant(__read_implies(y, __read_offset_bytes(y)/sizeof(float)/N == threadIdx.x)),
       __invariant(__write_implies(y, __write_offset_bytes(y)/sizeof(float)/N == threadIdx.x)),
       i<N; i++) {
    unsigned idx = (tid * N) + i;
    y[idx] = a * x[idx] + y[idx];
  }
}
