#include "hip/hip_runtime.h"
//--blockDim=[4,2] --gridDim=[2,2] -DWIDTH=8 -DHEIGHT=8 -DTILE_DIM=4 -DBLOCK_ROWS=2

// Example taken from transpose benchmark in the CUDA SDK (v5.0)
// Note fly in ointment with threadIdx.y invariant (should investigate)

#define WIDTH 8
#define HEIGHT 8
#define TILE_DIM 4
#define BLOCK_ROWS 2

__global__ void transpose(float *odata, float *idata, int width, int height) {
    __requires(width == WIDTH);
    __requires(height == HEIGHT);

    // additional preconditions that we check
    __assert(blockDim.x == TILE_DIM);
    __assert(blockDim.y == BLOCK_ROWS);
    __assert(width  == gridDim.x * TILE_DIM);
    __assert(height == gridDim.y * TILE_DIM);

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    int index_in  = xIndex + width * yIndex;
    int index_out = yIndex + height * xIndex;
  
    for (int i=0;
         __invariant(__mod_pow2(i, BLOCK_ROWS) == 0),
         __invariant(0 <= i),
         __invariant(i <= TILE_DIM),
         __invariant(__write_implies(odata, __write_offset_bytes(odata)/sizeof(float) / HEIGHT % TILE_DIM == threadIdx.x)),
         __invariant(__write_implies(odata, __write_offset_bytes(odata)/sizeof(float) % HEIGHT % TILE_DIM % BLOCK_ROWS == threadIdx.y)),
         __invariant(__write_implies(odata, __write_offset_bytes(odata)/sizeof(float) / HEIGHT / TILE_DIM == blockIdx.x)),
         __invariant(__write_implies(odata, __write_offset_bytes(odata)/sizeof(float) % HEIGHT / TILE_DIM == blockIdx.y)),
         i<TILE_DIM; i+=BLOCK_ROWS)
    {
        odata[index_out+i] = idata[index_in+i*width];
    }
}
